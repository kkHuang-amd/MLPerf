#include "hip/hip_runtime.h"
#include "dot_based_interact_common.cuh"

using namespace dlrm_dot;

template <uint THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__
  void dotBasedInteractNoWmmaFwdKernelNonAligned(const half *__restrict input,
                                                 half *__restrict output,
                                                 uint batch_size,
                                                 uint num_rows,
                                                 uint num_cols,
                                                 uint input_size,
                                                 uint output_size,
                                                 uint interaction_output_size) {
  extern __shared__ half smem_f16_fwd_na[];
  half *smem_in = &smem_f16_fwd_na[0];

  uint input_batch_offset = blockIdx.x * input_size;
  const half *gmem_in = &input[input_batch_offset];

  uint output_batch_offset = blockIdx.x * output_size;
  half *gmem_out_bottom_mlp = &output[output_batch_offset];
  half *gmem_out_interaction = &output[output_batch_offset + num_cols];

  // Load the input - one sample per block
  for (uint idx = threadIdx.x; idx < input_size; idx += blockDim.x) {
    smem_in[idx] = gmem_in[idx];
  }
  __syncthreads();

  // Copy bottom MLP output to output
  for (uint idx = threadIdx.x; idx < num_cols; idx += blockDim.x) {
    gmem_out_bottom_mlp[idx] = smem_in[idx];
  }

  for (uint idx = threadIdx.x; idx < (interaction_output_size); idx += blockDim.x) {
    uint elems_per_row = 1;
    uint index = idx;
    while (index >= elems_per_row) {
      index -= elems_per_row;
      elems_per_row++;
    }
    uint target_row = elems_per_row;
    uint target_col = index;

    half sum = __float2half(0);
    for (uint i = 0; i < num_cols; i++) {
      half tmp1 = smem_in[target_row * num_cols + i];
      half tmp2 = smem_in[target_col * num_cols + i];
      sum = __hfma(tmp1, tmp2, sum);
    }

    gmem_out_interaction[idx] = sum;
  }

  gmem_out_interaction[interaction_output_size] = __float2half(0);
}

template <uint THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__
  void dotBasedInteractNoWmmaFwdKernel(const half *__restrict input,
                                       half *__restrict output,
                                       uint batch_size,
                                       uint num_rows,
                                       uint num_cols,
                                       uint input_size,
                                       uint output_size,
                                       uint interaction_output_size) {
  extern __shared__ half smem_f16_fwd[];
  half *smem_in = &smem_f16_fwd[0];

  uint input_batch_offset = blockIdx.x * input_size;
  const half *gmem_in = &input[input_batch_offset];

  uint output_batch_offset = blockIdx.x * output_size;
  half *gmem_out_bottom_mlp = &output[output_batch_offset];
  half *gmem_out_interaction = &output[output_batch_offset + num_cols];

  // Load the input - one sample per block
  uint input_size_half4 = input_size >> 2;
  for (uint idx = threadIdx.x; idx < input_size_half4; idx += blockDim.x) {
    ((half4 *)smem_in)[idx] = ((half4 *)gmem_in)[idx];
  }
  __syncthreads();

  // Copy bottom MLP output to output
  uint btm_mlp_out_size_half4 = num_cols >> 2;
  for (uint idx = threadIdx.x; idx < btm_mlp_out_size_half4; idx += blockDim.x) {
    ((half4 *)gmem_out_bottom_mlp)[idx] = ((half4 *)smem_in)[idx];
  }

  for (uint idx = threadIdx.x; idx < (interaction_output_size); idx += blockDim.x) {
    uint elems_per_row = 1;
    uint index = idx;
    while (index >= elems_per_row) {
      index -= elems_per_row;
      elems_per_row++;
    }
    uint target_row = elems_per_row;
    uint target_col = index;

    half4 sum;
    sum.vals[0] = __float2half2_rn(0);
    sum.vals[1] = __float2half2_rn(0);
    uint num_cols_half4 = num_cols >> 2;
    for (uint i = 0; i < num_cols_half4; i++) {
      half4 tmp1 = ((half4 *)smem_in)[target_row * num_cols_half4 + i];
      half4 tmp2 = ((half4 *)smem_in)[target_col * num_cols_half4 + i];
      sum.vals[0] = __hfma2(tmp1.vals[0], tmp2.vals[0], sum.vals[0]);
      sum.vals[1] = __hfma2(tmp1.vals[1], tmp2.vals[1], sum.vals[1]);
    }

    half sum_val0 = __hadd(__low2half(sum.vals[0]), __high2half(sum.vals[0]));
    half sum_val1 = __hadd(__low2half(sum.vals[1]), __high2half(sum.vals[1]));
    gmem_out_interaction[idx] = __hadd(sum_val0, sum_val1);
  }

  gmem_out_interaction[interaction_output_size] = __float2half(0);
}

template <uint THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__
  void dotBasedInteractNoWmmaFwdKernelNonAligned(const float *__restrict input,
                                                 float *__restrict output,
                                                 uint batch_size,
                                                 uint num_rows,
                                                 uint num_cols,
                                                 uint input_size,
                                                 uint output_size,
                                                 uint interaction_output_size) {
  extern __shared__ float smem_f32_fwd[];
  float *smem_in = &smem_f32_fwd[0];

  uint input_batch_offset = blockIdx.x * input_size;
  const float *gmem_in = &input[input_batch_offset];

  uint output_batch_offset = blockIdx.x * output_size;
  float *gmem_out_bottom_mlp = &output[output_batch_offset];
  float *gmem_out_interaction = &output[output_batch_offset + num_cols];

  // Load the input - one sample per block
  for (uint idx = threadIdx.x; idx < input_size; idx += blockDim.x) {
    smem_in[idx] = gmem_in[idx];
  }
  __syncthreads();

  // Copy bottom MLP output to output
  for (uint idx = threadIdx.x; idx < num_cols; idx += blockDim.x) {
    gmem_out_bottom_mlp[idx] = smem_in[idx];
  }

  for (uint idx = threadIdx.x; idx < (interaction_output_size); idx += blockDim.x) {
    uint elems_per_row = 1;
    uint index = idx;
    while (index >= elems_per_row) {
      index -= elems_per_row;
      elems_per_row++;
    }
    uint target_row = elems_per_row;
    uint target_col = index;

    float sum = 0;
    for (uint i = 0; i < num_cols; i++) {
      float tmp1 = smem_in[target_row * num_cols + i];
      float tmp2 = smem_in[target_col * num_cols + i];
      sum = fmaf(tmp1, tmp2, sum);
    }

    gmem_out_interaction[idx] = sum;
  }

  gmem_out_interaction[interaction_output_size] = 0;
}

template <uint THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__
  void dotBasedInteractNoWmmaFwdKernel(const float *__restrict input,
                                       float *__restrict output,
                                       uint batch_size,
                                       uint num_rows,
                                       uint num_cols,
                                       uint input_size,
                                       uint output_size,
                                       uint interaction_output_size) {
  extern __shared__ float smem_f32_fwd[];
  float *smem_in = &smem_f32_fwd[0];

  uint input_batch_offset = blockIdx.x * input_size;
  const float *gmem_in = &input[input_batch_offset];

  uint output_batch_offset = blockIdx.x * output_size;
  float *gmem_out_bottom_mlp = &output[output_batch_offset];
  float *gmem_out_interaction = &output[output_batch_offset + num_cols];

  // Load the input - one sample per block
  uint input_size_float4 = input_size >> 2;
  for (uint idx = threadIdx.x; idx < input_size_float4; idx += blockDim.x) {
    ((float4 *)smem_in)[idx] = ((float4 *)gmem_in)[idx];
  }
  __syncthreads();

  // Copy bottom MLP output to output
  uint btm_mlp_out_size_float4 = num_cols >> 2;
  for (uint idx = threadIdx.x; idx < btm_mlp_out_size_float4; idx += blockDim.x) {
    ((float4 *)gmem_out_bottom_mlp)[idx] = ((float4 *)smem_in)[idx];
  }

  for (uint idx = threadIdx.x; idx < (interaction_output_size); idx += blockDim.x) {
    uint elems_per_row = 1;
    uint index = idx;
    while (index >= elems_per_row) {
      index -= elems_per_row;
      elems_per_row++;
    }
    uint target_row = elems_per_row;
    uint target_col = index;

    float4 sum;
    sum.x = 0;
    sum.y = 0;
    sum.z = 0;
    sum.w = 0;
    uint num_cols_float4 = num_cols >> 2;
    for (uint i = 0; i < num_cols_float4; i++) {
      float4 tmp1 = ((float4 *)smem_in)[target_row * num_cols_float4 + i];
      float4 tmp2 = ((float4 *)smem_in)[target_col * num_cols_float4 + i];
      sum.x = fmaf(tmp1.x, tmp2.x, sum.x);
      sum.y = fmaf(tmp1.y, tmp2.y, sum.y);
      sum.z = fmaf(tmp1.z, tmp2.z, sum.z);
      sum.w = fmaf(tmp1.w, tmp2.w, sum.w);
    }

    gmem_out_interaction[idx] = sum.x + sum.y + sum.z + sum.w;
  }

  gmem_out_interaction[interaction_output_size] = 0;
}

template <uint THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__
  void dotBasedInteractNoWmmaBwdKernelNonAligned(const half *__restrict input,
                                                 const half *__restrict upstream_grad,
                                                 half *__restrict grad,
                                                 half *__restrict bottom_mlp_grad,
                                                 uint batch_size,
                                                 uint num_rows,
                                                 uint num_cols,
                                                 uint input_size,
                                                 uint ugrad_size,
                                                 uint interaction_ugrad_size) {
  extern __shared__ half smem_f16_bwd[];
  half *smem_in = &smem_f16_bwd[0];
  half *smem_interaction_ugrad = &smem_f16_bwd[input_size];

  // Input
  uint input_batch_offset = blockIdx.x * input_size;
  const half *gmem_in = &input[input_batch_offset];

  // Gradient
  const uint &grad_batch_offset = input_batch_offset;
  half *gmem_mlp_grad = &bottom_mlp_grad[blockIdx.x * num_cols];
  half *gmem_interaction_grad = &grad[grad_batch_offset];

  // Upstream Gradient
  uint upstream_grad_batch_offset = blockIdx.x * ugrad_size;
  const half *gmem_mlp_ugrad = &upstream_grad[upstream_grad_batch_offset];
  const half *gmem_interaction_ugrad = &upstream_grad[upstream_grad_batch_offset + num_cols];

  // input -> shared memory
  for (uint idx = threadIdx.x; idx < input_size; idx += blockDim.x) {
    smem_in[idx] = gmem_in[idx];
  }

  // Interaction Upstream Grad -> Shared Memory
  for (uint idx = threadIdx.x; idx < interaction_ugrad_size; idx += blockDim.x) {
    smem_interaction_ugrad[idx] = gmem_interaction_ugrad[idx];
  }
  __syncthreads();

  // Copy the upstream gradient w.r.t to mlp to it's corresponding memory location.
  for (uint idx = threadIdx.x; idx < num_cols; idx += blockDim.x) {
    gmem_mlp_grad[idx] = gmem_mlp_ugrad[idx];
  }

  for (uint idx = threadIdx.x; idx < num_cols; idx += blockDim.x) {
    size_t grad_idx = idx;
    for (uint row_idx = 0; row_idx < num_rows; row_idx++) {
      half sum = __float2half(0);
      size_t upstream_grad_offset = (row_idx * (row_idx - 1)) >> 1;
      for (int k = 0; k < row_idx; k++) {
        sum = __hfma(smem_in[k * num_cols + idx], smem_interaction_ugrad[upstream_grad_offset + k], sum);
      }
      for (int k = row_idx + 1; k < num_rows; k++) {
        upstream_grad_offset = (k * (k - 1)) >> 1;  // TODO: this can become a sum
        sum = __hfma(smem_in[k * num_cols + idx], smem_interaction_ugrad[upstream_grad_offset + row_idx], sum);
      }
      gmem_interaction_grad[grad_idx] = sum;
      grad_idx += num_cols;
    }
  }
}

template <uint THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__
  void dotBasedInteractNoWmmaBwdKernel(const half *__restrict input,
                                       const half *__restrict upstream_grad,
                                       half *__restrict grad,
                                       half *__restrict bottom_mlp_grad,
                                       uint batch_size,
                                       uint num_rows,
                                       uint num_cols,
                                       uint input_size,
                                       uint ugrad_size,
                                       uint interaction_ugrad_size) {
  extern __shared__ half smem_f16_bwd[];
  half *smem_in = &smem_f16_bwd[0];
  half *smem_interaction_ugrad = &smem_f16_bwd[input_size];

  // Input
  uint input_batch_offset = blockIdx.x * input_size;
  const half *gmem_in = &input[input_batch_offset];

  // Gradient
  const uint &grad_batch_offset = input_batch_offset;
  half *gmem_mlp_grad = &bottom_mlp_grad[blockIdx.x * num_cols];
  half *gmem_interaction_grad = &grad[grad_batch_offset];

  // Upstream Gradient
  uint upstream_grad_batch_offset = blockIdx.x * ugrad_size;
  const half *gmem_mlp_ugrad = &upstream_grad[upstream_grad_batch_offset];
  const half *gmem_interaction_ugrad = &upstream_grad[upstream_grad_batch_offset + num_cols];

  // input -> shared memory
  uint input_size_half4 = input_size >> 2;
  for (uint idx = threadIdx.x; idx < input_size_half4; idx += blockDim.x) {
    ((half4 *)smem_in)[idx] = ((half4 *)gmem_in)[idx];
  }

  // Interaction Upstream Grad -> Shared Memory
  uint upstream_grad_size_half4 = interaction_ugrad_size >> 2;
  for (uint idx = threadIdx.x; idx < upstream_grad_size_half4; idx += blockDim.x) {
    ((half4 *)smem_interaction_ugrad)[idx] = ((half4 *)gmem_interaction_ugrad)[idx];
  }

  uint vectorized_load_offset = (upstream_grad_size_half4 << 2);
  for (uint idx = vectorized_load_offset + threadIdx.x; idx < interaction_ugrad_size; idx += blockDim.x) {
    smem_interaction_ugrad[idx] = gmem_interaction_ugrad[idx];
  }
  __syncthreads();

  // Copy the upstream gradient w.r.t to mlp to it's corresponding memory location.
  for (uint idx = threadIdx.x; idx < (num_cols >> 2); idx += blockDim.x) {
    ((half4 *)gmem_mlp_grad)[idx] = ((half4 *)gmem_mlp_ugrad)[idx];
  }

  for (uint idx = threadIdx.x; idx < num_cols; idx += blockDim.x) {
    size_t grad_idx = idx;
    for (uint row_idx = 0; row_idx < num_rows; row_idx++) {
      half sum = __float2half(0);
      size_t upstream_grad_offset = (row_idx * (row_idx - 1)) >> 1;
      for (int k = 0; k < row_idx; k++) {
        sum = __hfma(smem_in[k * num_cols + idx], smem_interaction_ugrad[upstream_grad_offset + k], sum);
      }
      for (int k = row_idx + 1; k < num_rows; k++) {
        upstream_grad_offset = (k * (k - 1)) >> 1;  // TODO: this can become a sum
        sum = __hfma(smem_in[k * num_cols + idx], smem_interaction_ugrad[upstream_grad_offset + row_idx], sum);
      }
      gmem_interaction_grad[grad_idx] = sum;
      grad_idx += num_cols;
    }
  }
}

template <uint THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__
  void dotBasedInteractNoWmmaBwdKernelNonAligned(const float *__restrict input,
                                                 const float *__restrict upstream_grad,
                                                 float *__restrict grad,
                                                 float *__restrict bottom_mlp_grad,
                                                 uint batch_size,
                                                 uint num_rows,
                                                 uint num_cols,
                                                 uint input_size,
                                                 uint ugrad_size,
                                                 uint interaction_ugrad_size) {
  extern __shared__ float smem_f32_bwd_na[];
  float *smem_in = &smem_f32_bwd_na[0];
  float *smem_interaction_ugrad = &smem_f32_bwd_na[input_size];

  // Input
  uint input_batch_offset = blockIdx.x * input_size;
  const float *gmem_in = &input[input_batch_offset];

  // Gradient
  const uint &grad_batch_offset = input_batch_offset;
  float *gmem_mlp_grad = &bottom_mlp_grad[blockIdx.x * num_cols];
  float *gmem_interaction_grad = &grad[grad_batch_offset];

  // Upstream Gradient
  uint upstream_grad_batch_offset = blockIdx.x * ugrad_size;
  const float *gmem_mlp_ugrad = &upstream_grad[upstream_grad_batch_offset];
  const float *gmem_interaction_ugrad = &upstream_grad[upstream_grad_batch_offset + num_cols];

  // input -> shared memory
  for (uint idx = threadIdx.x; idx < input_size; idx += blockDim.x) {
    smem_in[idx] = gmem_in[idx];
  }

  // Interaction Upstream Grad -> Shared Memory
  for (uint idx = threadIdx.x; idx < interaction_ugrad_size; idx += blockDim.x) {
    smem_interaction_ugrad[idx] = gmem_interaction_ugrad[idx];
  }
  __syncthreads();

  // Copy the upstream gradient w.r.t to mlp to it's corresponding memory location.
  for (uint idx = threadIdx.x; idx < num_cols; idx += blockDim.x) {
    gmem_mlp_grad[idx] = gmem_mlp_ugrad[idx];
  }

  for (uint idx = threadIdx.x; idx < num_cols; idx += blockDim.x) {
    size_t grad_idx = idx;
    for (uint row_idx = 0; row_idx < num_rows; row_idx++) {
      float sum = 0;
      size_t upstream_grad_offset = (row_idx * (row_idx - 1)) >> 1;
      for (int k = 0; k < row_idx; k++) {
        sum = fmaf(smem_in[k * num_cols + idx], smem_interaction_ugrad[upstream_grad_offset + k], sum);
      }
      for (int k = row_idx + 1; k < num_rows; k++) {
        upstream_grad_offset = (k * (k - 1)) >> 1;  // TODO: this can become a sum
        sum = fmaf(smem_in[k * num_cols + idx], smem_interaction_ugrad[upstream_grad_offset + row_idx], sum);
      }
      gmem_interaction_grad[grad_idx] = sum;
      grad_idx += num_cols;
    }
  }
}

template <uint THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__
  void dotBasedInteractNoWmmaBwdKernel(const float *__restrict input,
                                       const float *__restrict upstream_grad,
                                       float *__restrict grad,
                                       float *__restrict bottom_mlp_grad,
                                       uint batch_size,
                                       uint num_rows,
                                       uint num_cols,
                                       uint input_size,
                                       uint ugrad_size,
                                       uint interaction_ugrad_size) {
  extern __shared__ float smem_f32_bwd[];
  float *smem_in = &smem_f32_bwd[0];
  float *smem_interaction_ugrad = &smem_f32_bwd[input_size];

  // Input
  uint input_batch_offset = blockIdx.x * input_size;
  const float *gmem_in = &input[input_batch_offset];

  // Gradient
  const uint &grad_batch_offset = input_batch_offset;
  float *gmem_mlp_grad = &bottom_mlp_grad[blockIdx.x * num_cols];
  float *gmem_interaction_grad = &grad[grad_batch_offset];

  // Upstream Gradient
  uint upstream_grad_batch_offset = blockIdx.x * ugrad_size;
  const float *gmem_mlp_ugrad = &upstream_grad[upstream_grad_batch_offset];
  const float *gmem_interaction_ugrad = &upstream_grad[upstream_grad_batch_offset + num_cols];

  // input -> shared memory
  uint input_size_float4 = input_size >> 2;
  for (uint idx = threadIdx.x; idx < input_size_float4; idx += blockDim.x) {
    ((float4 *)smem_in)[idx] = ((float4 *)gmem_in)[idx];
  }

  // Interaction Upstream Grad -> Shared Memory
  uint upstream_grad_size_float4 = interaction_ugrad_size >> 2;
  for (uint idx = threadIdx.x; idx < upstream_grad_size_float4; idx += blockDim.x) {
    ((float4 *)smem_interaction_ugrad)[idx] = ((float4 *)gmem_interaction_ugrad)[idx];
  }

  uint vectorized_load_offset = (upstream_grad_size_float4 << 2);
  for (uint idx = vectorized_load_offset + threadIdx.x; idx < interaction_ugrad_size; idx += blockDim.x) {
    smem_interaction_ugrad[idx] = gmem_interaction_ugrad[idx];
  }
  __syncthreads();

  // Copy the upstream gradient w.r.t to mlp to it's corresponding memory location.
  for (uint idx = threadIdx.x; idx < (num_cols >> 2); idx += blockDim.x) {
    ((float4 *)gmem_mlp_grad)[idx] = ((float4 *)gmem_mlp_ugrad)[idx];
  }

  for (uint idx = threadIdx.x; idx < num_cols; idx += blockDim.x) {
    size_t grad_idx = idx;
    for (uint row_idx = 0; row_idx < num_rows; row_idx++) {
      float sum = 0;
      size_t upstream_grad_offset = (row_idx * (row_idx - 1)) >> 1;
      for (int k = 0; k < row_idx; k++) {
        sum = fmaf(smem_in[k * num_cols + idx], smem_interaction_ugrad[upstream_grad_offset + k], sum);
      }
      for (int k = row_idx + 1; k < num_rows; k++) {
        upstream_grad_offset = (k * (k - 1)) >> 1;  // TODO: this can become a sum
        sum = fmaf(smem_in[k * num_cols + idx], smem_interaction_ugrad[upstream_grad_offset + row_idx], sum);
      }
      gmem_interaction_grad[grad_idx] = sum;
      grad_idx += num_cols;
    }
  }
}

inline void dotBasedInteractNoWmmaFwd(const void *input,
                                      const void *bottom_mlp_output,
                                      const void *output,
                                      uint batch_size,
                                      uint num_rows,
                                      uint num_cols,
                                      uint pad,
                                      bool amp_train) {
  const uint kNumThreads = 128;
  uint num_blocks = batch_size;

  // Output
  uint interaction_output_size = (num_rows * (num_rows - 1)) >> 1;
  uint output_size = num_cols + interaction_output_size + pad;

  // Input
  uint input_size = num_rows * num_cols;

  uint shared_mem_size_elems = input_size;
  uint shared_mem_size_bytes = shared_mem_size_elems << 2;  // F32 Kernel

  bool float4_predicate = !((num_cols & 3) || (output_size & 3));

  if (float4_predicate) {
    if (amp_train) {
      dotBasedInteractNoWmmaFwdKernel<kNumThreads>
          <<<num_blocks, kNumThreads, shared_mem_size_bytes>>>((const half *)input,
                                                               (half *)output,
                                                               batch_size,
                                                               num_rows,
                                                               num_cols,
                                                               input_size,
                                                               output_size,
                                                               interaction_output_size);
    }
    else {
      dotBasedInteractNoWmmaFwdKernel<kNumThreads>
          <<<num_blocks, kNumThreads, shared_mem_size_bytes>>>((const float *)input,
                                                               (float *)output,
                                                               batch_size,
                                                               num_rows,
                                                               num_cols,
                                                               input_size,
                                                               output_size,
                                                               interaction_output_size);
    }
  } else {
    if (amp_train) {
      dotBasedInteractNoWmmaFwdKernelNonAligned<kNumThreads>
          <<<num_blocks, kNumThreads, shared_mem_size_bytes>>>((const half *)input,
                                                               (half *)output,
                                                               batch_size,
                                                               num_rows,
                                                               num_cols,
                                                               input_size,
                                                               output_size,
                                                               interaction_output_size);
    }
    else {
      dotBasedInteractNoWmmaFwdKernelNonAligned<kNumThreads>
          <<<num_blocks, kNumThreads, shared_mem_size_bytes>>>((const float *)input,
                                                               (float *)output,
                                                               batch_size,
                                                               num_rows,
                                                               num_cols,
                                                               input_size,
                                                               output_size,
                                                               interaction_output_size);
    }
  }
}

inline void dotBasedInteractNoWmmaBwd(const void *input,
                                      const void *upstream_grad,
                                      void *grad,
                                      void *bottom_mlp_grad,
                                      uint batch_size,
                                      uint num_rows,
                                      uint num_cols,
                                      uint pad,
                                      bool amp_train) {
  const uint kNumThreads = 128;

  uint num_blocks = batch_size;

  uint input_size = num_rows * num_cols;

  // 1D ugrad size
  uint interaction_ugrad_size = num_rows * (num_rows - 1) >> 1;
  uint interaction_ugrad_size_with_padding = interaction_ugrad_size + pad;
  uint ugrad_size = num_cols + interaction_ugrad_size_with_padding;

  // input space + upstream grad space
  uint smem_size_elems = input_size + interaction_ugrad_size;
  uint smem_size_bytes = smem_size_elems << 2;  // F32 Kernel

  bool float4_predicate = !((interaction_ugrad_size_with_padding & 3) || (num_cols & 3));
  if (float4_predicate) {
    if (amp_train) {
      dotBasedInteractNoWmmaBwdKernel<kNumThreads>
          <<<num_blocks, kNumThreads, smem_size_bytes>>>((const half *)input,
                                                         (const half *)upstream_grad,
                                                         (half *)grad,
                                                         (half *)bottom_mlp_grad,
                                                         batch_size,
                                                         num_rows,
                                                         num_cols,
                                                         input_size,
                                                         ugrad_size,
                                                         interaction_ugrad_size);
    }
    else {
      dotBasedInteractNoWmmaBwdKernel<kNumThreads>
          <<<num_blocks, kNumThreads, smem_size_bytes>>>((const float *)input,
                                                         (const float *)upstream_grad,
                                                         (float *)grad,
                                                         (float *)bottom_mlp_grad,
                                                         batch_size,
                                                         num_rows,
                                                         num_cols,
                                                         input_size,
                                                         ugrad_size,
                                                         interaction_ugrad_size);
    }
  } else {
    if (amp_train) {
      dotBasedInteractNoWmmaBwdKernelNonAligned<kNumThreads>
          <<<num_blocks, kNumThreads, smem_size_bytes>>>((const half *)input,
                                                         (const half *)upstream_grad,
                                                         (half *)grad,
                                                         (half *)bottom_mlp_grad,
                                                         batch_size,
                                                         num_rows,
                                                         num_cols,
                                                         input_size,
                                                         ugrad_size,
                                                         interaction_ugrad_size);
    }
    else {
      dotBasedInteractNoWmmaBwdKernelNonAligned<kNumThreads>
          <<<num_blocks, kNumThreads, smem_size_bytes>>>((const float *)input,
                                                         (const float *)upstream_grad,
                                                         (float *)grad,
                                                         (float *)bottom_mlp_grad,
                                                         batch_size,
                                                         num_rows,
                                                         num_cols,
                                                         input_size,
                                                         ugrad_size,
                                                         interaction_ugrad_size);
    }
  }
}
