
#include <hip/hip_runtime.h>
#ifdef USE_WMMA
#ifdef __HIP_PLATFORM_HCC__
#include <rocwmma/rocwmma.hpp>
#else
#include <mma.h>
#endif

#include "dot_based_interact_common.cuh"

#ifdef __HIP_PLATFORM_HCC__
using namespace rocwmma;
#else
using namespace nvcuda;
using namespace wmma;
#endif
using namespace dlrm_dot;

__device__ inline void store(__half *dst, float *src) {
  *dst = __float2half(*src);
}

__device__ inline void store(__half *dst, const float src) {
  *dst = __float2half(src);
}

__device__ inline void store(float *dst, float *src) {
  *dst = *src;
}

__device__ inline void store(float *dst, const float src) {
  *dst = src;
}

__device__ static inline void syncwarp() {
#ifdef __HIP_PLATFORM_HCC__
  __builtin_amdgcn_wave_barrier();
#else
  __syncwarp();
#endif
}

template <uint TILE_DIM,
          uint M_BLOCKS,
          uint SMEM_STRIDE,
          uint SMEM_STRIDE_ACC,
          typename T>
__device__ inline void bmmTrilPadFwdKernel(T *shmem,
                                           T *gmem_output,
                                           uint num_rows,
                                           uint num_cols,
                                           uint smem_rows_per_warp,
                                           uint output_size,
                                           uint num_col_steps,
                                           uint pad,
                                           int lane_id) {
  fragment<accumulator, TILE_DIM, TILE_DIM, TILE_DIM, float> acc[M_BLOCKS][M_BLOCKS];

  for (int i = 0; i < M_BLOCKS; i++) {
    for (int j = 0; j < M_BLOCKS; j++) {
      fill_fragment(acc[i][j], 0.0f);
    }
  }

  for (int k_step = 0; k_step < num_col_steps; k_step++) {
    fragment<matrix_a, TILE_DIM, TILE_DIM, TILE_DIM, T, row_major> a[M_BLOCKS];
    fragment<matrix_b, TILE_DIM, TILE_DIM, TILE_DIM, T, col_major> b[M_BLOCKS];
    for (int j = 0; j < M_BLOCKS; j++) {
      int base_row = (j < M_BLOCKS - 1) ? j * 16 : smem_rows_per_warp - 16;
      const T *tile_ptr = shmem + (base_row * SMEM_STRIDE + k_step * 16);
      load_matrix_sync(a[j], tile_ptr, SMEM_STRIDE);
      load_matrix_sync(b[j], tile_ptr, SMEM_STRIDE);
    }
    for (int i = 0; i < M_BLOCKS; i++) {
      for (int j = 0; j < M_BLOCKS; j++) {
        mma_sync(acc[i][j], a[i], b[j], acc[i][j]);
      }
    }
  }
  float *shmem_store = reinterpret_cast<float *>(shmem);
  for (int i = 0; i < M_BLOCKS; i++) {
    for (int j = 0; j < M_BLOCKS; j++) {
      float *tile_ptr = shmem_store + (i * 16 * SMEM_STRIDE_ACC + j * 16);
      store_matrix_sync(tile_ptr, acc[i][j], SMEM_STRIDE_ACC, mem_row_major);
    }
  }

  T *gmem_interact_output = gmem_output + num_cols;
  int lastRowBlockOffset = M_BLOCKS * 16 - smem_rows_per_warp;
  int srcLine = 0;
  for (int i = 0; i < num_rows; ++i, ++srcLine) {
    if (i == ((M_BLOCKS - 1) * 16)) {
      srcLine += lastRowBlockOffset;
    }
    if (lane_id < i) {
      uint offset = (i * (i - 1)) >> 1;
      store(gmem_interact_output + offset + lane_id, shmem_store + srcLine * SMEM_STRIDE_ACC + lane_id);
    }
  }

  // Padding
  if (lane_id < pad) {
    store(gmem_output + lane_id + output_size - 1, 0.0f);
  }
}

template <typename T>
__device__ inline void trilBwdKernel(T *smem_in,
                                     T *smem_temp,
                                     T zero,
                                     uint num_rows,
                                     uint num_rows_after_padding,
                                     uint interaction_ugrad_2D_stride,
                                     uint lane_id) {
  if (lane_id < num_rows_after_padding) {
    uint ugrad_flat_index = ((lane_id * (lane_id - 1)) >> 1);
    uint ugrad_offset_1 = lane_id * interaction_ugrad_2D_stride;
    for (uint row = 0; row < num_rows; row++) {
      T ugrad_val = zero;
      if (row < lane_id && lane_id < num_rows) {
        ugrad_val = smem_in[ugrad_flat_index + row];
        smem_temp[ugrad_offset_1 + row] = ugrad_val;
      }
      if (row <= lane_id && lane_id < num_rows_after_padding) {
        smem_temp[row * interaction_ugrad_2D_stride + lane_id] = ugrad_val;
      }
    }
    for (uint row = num_rows; row < num_rows_after_padding; row++) {
      smem_temp[row * interaction_ugrad_2D_stride + lane_id] = zero;
    }
  }
}

template <uint TILE_DIM,
          uint ROW_TILES_PER_STEP,
          uint TILE_DIM_LOG_2,
          typename T>
__device__ inline void bmmBwdKernel(T *smem_in,
                                    T *smem_temp,
                                    float *smem_out,
                                    T *gmem_grad,
                                    uint num_rows,
                                    uint num_cols,
                                    uint num_col_steps,
                                    uint input_stride,
                                    uint interaction_ugrad_2D_stride,
                                    uint lane_id) {
  fragment<matrix_a, TILE_DIM, TILE_DIM, TILE_DIM, T, row_major> a[ROW_TILES_PER_STEP]
                                                                                       [ROW_TILES_PER_STEP];
  for (uint i = 0; i < ROW_TILES_PER_STEP; i++) {
    for (uint j = 0; j < ROW_TILES_PER_STEP; j++) {
      const T *tile_ptr = smem_temp + ((i * interaction_ugrad_2D_stride + j) << TILE_DIM_LOG_2);
      load_matrix_sync(a[i][j], tile_ptr, interaction_ugrad_2D_stride);
    }
  }

  fragment<accumulator, TILE_DIM, TILE_DIM, TILE_DIM, float> acc[ROW_TILES_PER_STEP];
  fragment<matrix_b, TILE_DIM, TILE_DIM, TILE_DIM, T, row_major> b[ROW_TILES_PER_STEP];
  for (int col_step = 0; col_step < num_col_steps; col_step++) {
    for (uint i = 0; i < ROW_TILES_PER_STEP; i++) {
      const T *tile_ptr = smem_in + ((i * input_stride + col_step) << TILE_DIM_LOG_2);
      fill_fragment(acc[i], 0.0f);
      load_matrix_sync(b[i], tile_ptr, input_stride);
    }
    for (uint i = 0; i < ROW_TILES_PER_STEP; i++) {
      for (uint j = 0; j < ROW_TILES_PER_STEP; j++) {
        mma_sync(acc[i], a[i][j], b[j], acc[i]);
      }
    }
    for (uint i = 0; i < ROW_TILES_PER_STEP; i++) {
      float *tile_ptr = smem_out + i * TILE_DIM * TILE_DIM;
      store_matrix_sync(tile_ptr, acc[i], TILE_DIM, mem_row_major);
    }

    syncwarp();

    uint gmem_grad_col = (col_step << TILE_DIM_LOG_2) + lane_id;
    if (gmem_grad_col < num_cols) {
      for (uint i = 0; i < num_rows; i++) {
        store(&gmem_grad[i * num_cols + gmem_grad_col], &smem_out[(i << TILE_DIM_LOG_2) + lane_id]);
      }
    }
  }
}

template <uint WARPS_PER_BLOCK,
          uint THREADBLOCK_SIZE,
          uint M_BLOCKS,
          uint K_BLOCKS,
          uint SMEM_STRIDE,
          uint SMEM_STRIDE_ACC,
          uint WARP_SIZE,
          uint WARP_SIZE_LOG_2,
          uint TILE_DIM,
          uint TILE_DIM_LOG_2>
__launch_bounds__(THREADBLOCK_SIZE) __global__
    void dotBasedInteractFwdKernelNonAligned(const __half *__restrict input,
                                             __half *__restrict output,
                                             uint batch_size,
                                             uint num_rows,
                                             uint num_cols,
                                             uint num_rows_after_padding,
                                             uint num_cols_after_padding,
                                             uint smem_elems_per_warp,
                                             uint smem_rows_per_warp,
                                             uint output_size,
                                             uint num_row_steps,
                                             uint num_col_steps,
                                             uint pad) {
  uint warp_id = (threadIdx.x >> WARP_SIZE_LOG_2);
  int sample_id = blockIdx.x * WARPS_PER_BLOCK + warp_id;
  if (sample_id >= batch_size) {
    return;
  }
  int lane_id = threadIdx.x & (WARP_SIZE - 1);

  extern __shared__ half shmem_fwd_na_h[];
  half *shmem = shmem_fwd_na_h + (warp_id * smem_elems_per_warp);

  const half *sample_input = input + num_rows * num_cols * sample_id;
  for (uint i = 0; i < num_rows; ++i, sample_input += num_cols) {
    for (uint idx = lane_id; idx < num_cols; idx += WARP_SIZE) {
      (shmem + i * SMEM_STRIDE)[idx] = sample_input[idx];
    }
  }

  uint idx = lane_id + num_cols;
  if (idx < num_cols_after_padding) {
    for (int i = 0; i < num_rows; ++i) {
      (shmem + i * SMEM_STRIDE)[idx] = __float2half(0);
    }
  }

  half4 zeros;
#ifdef __HIP_PLATFORM_HCC__
  zeros.vals[0] = __float2half2_rn(0);
  zeros.vals[1] = __float2half2_rn(0);
#else
  half4 zeros;
  zeros.vals[0].x = __float2half(0);
  zeros.vals[0].y = __float2half(0);
  zeros.vals[1].x = __float2half(0);
  zeros.vals[1].y = __float2half(0);
#endif
  if (lane_id < (num_cols_after_padding >> 2)) {
    for (int i = num_rows; i < num_rows_after_padding; i++) {
      ((half4 *)(shmem + i * SMEM_STRIDE))[lane_id] = zeros;
    }
  }

  syncwarp();

  half *gmem_output = output + output_size * sample_id;
  for (uint idx = lane_id; idx < num_cols; idx += WARP_SIZE) {
    gmem_output[idx] = shmem[idx];
  }

  bmmTrilPadFwdKernel<TILE_DIM, M_BLOCKS, SMEM_STRIDE, SMEM_STRIDE_ACC, __half>(
      shmem, gmem_output, num_rows, num_cols, smem_rows_per_warp, output_size,
      num_col_steps, pad, lane_id);
}

template <uint WARPS_PER_BLOCK,
          uint THREADBLOCK_SIZE,
          uint M_BLOCKS,
          uint K_BLOCKS,
          uint SMEM_STRIDE,
          uint SMEM_STRIDE_ACC,
          uint WARP_SIZE,
          uint WARP_SIZE_LOG_2,
          uint TILE_DIM,
          uint TILE_DIM_LOG_2>
__launch_bounds__(THREADBLOCK_SIZE) __global__
  void dotBasedInteractFwdKernel(const __half *__restrict input,
                                 __half *__restrict output,
                                 uint batch_size,
                                 uint num_rows,
                                 uint num_cols,
                                 uint num_rows_after_padding,
                                 uint num_cols_after_padding,
                                 uint smem_elems_per_warp,
                                 uint smem_rows_per_warp,
                                 uint output_size,
                                 uint num_row_steps,
                                 uint num_col_steps,
                                 uint pad) {
  uint warp_id = (threadIdx.x >> WARP_SIZE_LOG_2);
  int sample_id = blockIdx.x * WARPS_PER_BLOCK + warp_id;
  if (sample_id >= batch_size) {
    return;
  }
  int lane_id = threadIdx.x & (WARP_SIZE - 1);

  extern __shared__ half shmem_fwd_h[];
  half *shmem = shmem_fwd_h + (warp_id * smem_elems_per_warp);

  const half *sample_input = input + num_rows * num_cols * sample_id;
  if (lane_id < (num_cols >> 2)) {
    for (int i = 0; i < num_rows; ++i, sample_input += num_cols) {
      ((float2 *)(shmem + i * SMEM_STRIDE))[lane_id] = ((float2 *)sample_input)[lane_id];
    }
  }

  uint idx = lane_id + num_cols;
  if (idx < num_cols_after_padding) {
    for (int i = 0; i < num_rows; ++i) {
      (shmem + i * SMEM_STRIDE)[idx] = __float2half(0);
    }
  }

  half4 zeros;
#ifdef __HIP_PLATFORM_HCC__
  zeros.vals[0] = __float2half2_rn(0);
  zeros.vals[1] = __float2half2_rn(0);
#else
  zeros.vals[0].x = __float2half(0);
  zeros.vals[0].y = __float2half(0);
  zeros.vals[1].x = __float2half(0);
  zeros.vals[1].y = __float2half(0);
#endif
  if (lane_id < (num_cols_after_padding >> 2)) {
    for (int i = num_rows; i < num_rows_after_padding; i++) {
      ((half4 *)(shmem + i * SMEM_STRIDE))[lane_id] = zeros;
    }
  }

  syncwarp();

  half *gmem_output = output + output_size * sample_id;
  if (lane_id < (num_cols >> 2)) {
    ((float2 *)gmem_output)[lane_id] = ((float2 *)shmem)[lane_id];
  }

  bmmTrilPadFwdKernel<TILE_DIM, M_BLOCKS, SMEM_STRIDE, SMEM_STRIDE_ACC, __half>(
      shmem, gmem_output, num_rows, num_cols, smem_rows_per_warp, output_size,
      num_col_steps, pad, lane_id);
}

#ifdef __HIP_PLATFORM_HCC__
template <uint WARPS_PER_BLOCK,
          uint THREADBLOCK_SIZE,
          uint M_BLOCKS,
          uint K_BLOCKS,
          uint SMEM_STRIDE,
          uint SMEM_STRIDE_ACC,
          uint WARP_SIZE,
          uint WARP_SIZE_LOG_2,
          uint TILE_DIM,
          uint TILE_DIM_LOG_2>
__launch_bounds__(THREADBLOCK_SIZE) __global__
  void dotBasedInteractFwdKernelNonAligned(const float *__restrict input,
                                           float *__restrict output,
                                           uint batch_size,
                                           uint num_rows,
                                           uint num_cols,
                                           uint num_rows_after_padding,
                                           uint num_cols_after_padding,
                                           uint smem_elems_per_warp,
                                           uint smem_rows_per_warp,
                                           uint output_size,
                                           uint num_row_steps,
                                           uint num_col_steps,
                                           uint pad) {
  uint warp_id = (threadIdx.x >> WARP_SIZE_LOG_2);
  int sample_id = blockIdx.x * WARPS_PER_BLOCK + warp_id;
  if (sample_id >= batch_size) {
    return;
  }
  int lane_id = threadIdx.x & (WARP_SIZE - 1);

  extern __shared__ float shmem_fwd_na_f[];
  float *shmem = shmem_fwd_na_f + (warp_id * smem_elems_per_warp);

  const float *sample_input = input + num_rows * num_cols * sample_id;
  for (uint i = 0; i < num_rows; ++i, sample_input += num_cols) {
    for (uint idx = lane_id; idx < num_cols; idx += WARP_SIZE) {
      (shmem + i * SMEM_STRIDE)[idx] = sample_input[idx];
    }
  }

  uint idx = lane_id + num_cols;
  if (idx < num_cols_after_padding) {
    for (int i = 0; i < num_rows; ++i) {
      (shmem + i * SMEM_STRIDE)[idx] = 0;
    }
  }

  for (int i = num_rows; i < num_rows_after_padding; i++) {
    for (uint idx = lane_id; idx < num_cols; idx += WARP_SIZE) {
      (shmem + i * SMEM_STRIDE)[idx] = 0;
    }
  }

  __syncthreads();

  float *gmem_output = output + output_size * sample_id;
  for (uint idx = lane_id; idx < num_cols; idx += WARP_SIZE) {
    gmem_output[idx] = shmem[idx];
  }

  bmmTrilPadFwdKernel<TILE_DIM, M_BLOCKS, SMEM_STRIDE, SMEM_STRIDE_ACC, float>(
      shmem, gmem_output, num_rows, num_cols, smem_rows_per_warp, output_size,
      num_col_steps, pad, lane_id);
}

template <uint WARPS_PER_BLOCK,
          uint THREADBLOCK_SIZE,
          uint M_BLOCKS,
          uint K_BLOCKS,
          uint SMEM_STRIDE,
          uint SMEM_STRIDE_ACC,
          uint WARP_SIZE,
          uint WARP_SIZE_LOG_2,
          uint TILE_DIM,
          uint TILE_DIM_LOG_2>
__launch_bounds__(THREADBLOCK_SIZE) __global__
  void dotBasedInteractFwdKernel(const float *__restrict input,
                                 float *__restrict output,
                                 uint batch_size,
                                 uint num_rows,
                                 uint num_cols,
                                 uint num_rows_after_padding,
                                 uint num_cols_after_padding,
                                 uint smem_elems_per_warp,
                                 uint smem_rows_per_warp,
                                 uint output_size,
                                 uint num_row_steps,
                                 uint num_col_steps,
                                 uint pad) {
  uint warp_id = (threadIdx.x >> WARP_SIZE_LOG_2);
  int sample_id = blockIdx.x * WARPS_PER_BLOCK + warp_id;
  if (sample_id >= batch_size) {
    return;
  }
  int lane_id = threadIdx.x & (WARP_SIZE - 1);

  extern __shared__ float shmem_fwd_f[];
  float *shmem = shmem_fwd_f + (warp_id * smem_elems_per_warp);

  const float *sample_input = input + num_rows * num_cols * sample_id;
  if (lane_id < (num_cols >> 1)) {
    for (int i = 0; i < num_rows; ++i, sample_input += num_cols) {
      ((float2 *)(shmem + i * SMEM_STRIDE))[lane_id] = ((float2 *)sample_input)[lane_id];
    }
  }

  uint idx = lane_id + num_cols;
  if (idx < num_cols_after_padding) {
    for (int i = 0; i < num_rows; ++i) {
      (shmem + i * SMEM_STRIDE)[idx] = 0;
    }
  }

  for (int i = num_rows; i < num_rows_after_padding; i++) {
    for (uint idx = lane_id; idx < num_cols; idx += WARP_SIZE) {
      (shmem + i * SMEM_STRIDE)[idx] = 0;
    }
  }

  __syncthreads();

  float *gmem_output = output + output_size * sample_id;
  if (lane_id < (num_cols >> 1)) {
    ((float2 *)gmem_output)[lane_id] = ((float2 *)shmem)[lane_id];
  }

  bmmTrilPadFwdKernel<TILE_DIM, M_BLOCKS, SMEM_STRIDE, SMEM_STRIDE_ACC, float>(
      shmem, gmem_output, num_rows, num_cols, smem_rows_per_warp, output_size,
      num_col_steps, pad, lane_id);
}
#endif // __HIP_PLATFORM_HCC__

template <uint WARPS_PER_BLOCK,
          uint THREADBLOCK_SIZE,
          uint ROW_TILES_PER_STEP,
          uint COL_TILES_PER_STEP,
          uint WARP_SIZE,
          uint WARP_SIZE_LOG_2,
          uint TILE_DIM,
          uint TILE_DIM_LOG_2>
__launch_bounds__(THREADBLOCK_SIZE) __global__
    void dotBasedInteractBwdKernelNonAligned(const __half *__restrict input,
                                             const __half *__restrict upstream_grad,
                                             half *__restrict grad,
                                             half *__restrict bottom_mlp_grad,
                                             uint batch_size,
                                             uint num_rows,
                                             uint num_cols,
                                             uint num_rows_after_padding,
                                             uint num_cols_after_padding,
                                             uint sample_size,
                                             uint interaction_ugrad_size,
                                             uint interaction_ugrad_size_with_padding,
                                             uint interaction_ugrad_2D_size_elems,
                                             uint interaction_ugrad_2D_stride,
                                             uint input_size_elems,
                                             uint input_stride,
                                             uint num_row_steps,
                                             uint num_col_steps,
                                             uint row_tiles_per_step,
                                             uint shared_mem_per_warp_size_byte) {
  extern __shared__ half shmem_bwd_na_h[];
  uint warp_id = (threadIdx.x >> WARP_SIZE_LOG_2);
  uint sample_id = blockIdx.x * WARPS_PER_BLOCK + warp_id;
  if (sample_id >= batch_size) {
    return;
  }
  uint lane_id = threadIdx.x & (WARP_SIZE - 1);
  // ">> 1" to convert to half pointer
  uint smem_warp_offset = warp_id * (shared_mem_per_warp_size_byte >> 1);

  half *smem_in = shmem_bwd_na_h + smem_warp_offset;
  half *smem_temp = smem_in + input_size_elems;
  float *smem_out = reinterpret_cast<float *>(smem_temp);

  // Global memory pointers for the current sample
  // Input
  uint gmem_input_sample_offset = sample_id * sample_size;
  const half *gmem_input = &input[gmem_input_sample_offset];

  // Interaction Gradient
  const uint &gmem_grad_sample_offset = gmem_input_sample_offset;
  half *gmem_grad = &grad[gmem_grad_sample_offset];

  // Bottom MLP gradient
  half *gmem_mlp_grad = &bottom_mlp_grad[sample_id * num_cols];

  // Upstream gradient vector
  uint gmem_ugrad_sample_offset = sample_id * (num_cols + interaction_ugrad_size_with_padding);
  const half *gmem_ugrad = &upstream_grad[gmem_ugrad_sample_offset];

  // Upstream gradient vector for interactions
  const half *gmem_ugrad_interactions = &gmem_ugrad[num_cols];

  // upstream grad -> shared memory (place in input section temporarily)
#pragma unroll
  for (uint idx = lane_id; idx < interaction_ugrad_size; idx += WARP_SIZE) {
    smem_in[idx] = gmem_ugrad_interactions[idx];
  }

  syncwarp();

  // Form the 2D ugrad matrix.
  trilBwdKernel(smem_in, smem_temp, __float2half(0), num_rows, num_rows_after_padding,
      interaction_ugrad_2D_stride, lane_id);

  syncwarp();

  // Input -> Shared Memory

  for (uint row = 0; row < num_rows; row++) {
    half *smem_row_ptr = &smem_in[row * input_stride];
    const half *gmem_row_ptr = &gmem_input[row * num_cols];
    for (uint idx = lane_id; idx < num_cols; idx += WARP_SIZE) {
      smem_row_ptr[idx] = gmem_row_ptr[idx];
    }
    uint idx = lane_id + num_cols;
    if (idx < num_cols_after_padding) {
      smem_row_ptr[idx] = __float2half(0);
    }
  }

#pragma unroll 2
  for (uint row = num_rows; row < num_rows_after_padding; row++) {
    half *smem_row_ptr = &smem_in[row * input_stride];
    for (uint idx = lane_id; idx < num_cols_after_padding; idx += WARP_SIZE) {
      smem_row_ptr[idx] = __float2half(0);
    }
  }

  syncwarp();

  bmmBwdKernel<TILE_DIM, ROW_TILES_PER_STEP, TILE_DIM_LOG_2, __half>(
      smem_in, smem_temp, smem_out, gmem_grad, num_rows, num_cols, num_col_steps,
      input_stride, interaction_ugrad_2D_stride, lane_id);

  for (uint idx = lane_id; idx < num_cols; idx += WARP_SIZE) {
    gmem_mlp_grad[idx] = gmem_ugrad[idx];
  }
}

template <uint WARPS_PER_BLOCK,
          uint THREADBLOCK_SIZE,
          uint ROW_TILES_PER_STEP,
          uint COL_TILES_PER_STEP,
          uint WARP_SIZE,
          uint WARP_SIZE_LOG_2,
          uint TILE_DIM,
          uint TILE_DIM_LOG_2>
__launch_bounds__(THREADBLOCK_SIZE) __global__
  void dotBasedInteractBwdKernel(const __half *__restrict input,
                                 const __half *__restrict upstream_grad,
                                 half *__restrict grad,
                                 half *__restrict bottom_mlp_grad,
                                 uint batch_size,
                                 uint num_rows,
                                 uint num_cols,
                                 uint num_rows_after_padding,
                                 uint num_cols_after_padding,
                                 uint sample_size,
                                 uint interaction_ugrad_size,
                                 uint interaction_ugrad_size_with_padding,
                                 uint interaction_ugrad_2D_size_elems,
                                 uint interaction_ugrad_2D_stride,
                                 uint input_size_elems,
                                 uint input_stride,
                                 uint num_row_steps,
                                 uint num_col_steps,
                                 uint row_tiles_per_step,
                                 uint shared_mem_per_warp_size_byte) {
  extern __shared__ half shmem_bwd_h[];
  uint warp_id = (threadIdx.x >> WARP_SIZE_LOG_2);
  uint sample_id = blockIdx.x * WARPS_PER_BLOCK + warp_id;
  if (sample_id >= batch_size) {
    return;
  }
  uint lane_id = threadIdx.x & (WARP_SIZE - 1);
  // ">> 1" to convert to half pointer
  uint smem_warp_offset = warp_id * (shared_mem_per_warp_size_byte >> 1);

  half *smem_in = shmem_bwd_h + smem_warp_offset;
  half *smem_temp = smem_in + input_size_elems;
  float *smem_out = reinterpret_cast<float *>(smem_temp);

  // Global memory pointers for the current sample
  // Input
  uint gmem_input_sample_offset = sample_id * sample_size;
  const half *gmem_input = &input[gmem_input_sample_offset];

  // Interaction Gradient
  const uint &gmem_grad_sample_offset = gmem_input_sample_offset;
  half *gmem_grad = &grad[gmem_grad_sample_offset];

  // Bottom MLP gradient
  half *gmem_mlp_grad = &bottom_mlp_grad[sample_id * num_cols];

  // Upstream gradient vector
  uint gmem_ugrad_sample_offset = sample_id * (num_cols + interaction_ugrad_size_with_padding);
  const half *gmem_ugrad = &upstream_grad[gmem_ugrad_sample_offset];

  // Upstream gradient vector for interactions
  const half *gmem_ugrad_interactions = &gmem_ugrad[num_cols];

  // upstream grad -> shared memory (place in input section temporarily)
#pragma unroll
  for (uint idx = lane_id; idx < (interaction_ugrad_size >> 3); idx += WARP_SIZE) {
    ((float4 *)smem_in)[idx] = ((float4 *)gmem_ugrad_interactions)[idx];
  }
  uint offset = (interaction_ugrad_size >> 3) << 3;
  for (uint idx = lane_id + offset; idx < interaction_ugrad_size; idx += WARP_SIZE) {
    smem_in[idx] = gmem_ugrad_interactions[idx];
  }

  syncwarp();

  // Form the 2D ugrad matrix.
  trilBwdKernel(smem_in, smem_temp, __float2half(0), num_rows, num_rows_after_padding,
      interaction_ugrad_2D_stride, lane_id);

  syncwarp();

  // Input -> Shared Memory

  if (lane_id < (num_cols >> 2)) {
    for (uint row = 0; row < num_rows; row++) {
      half *smem_row_ptr = &smem_in[row * input_stride];
      const half *gmem_row_ptr = &gmem_input[row * num_cols];
      ((float2 *)smem_row_ptr)[lane_id] = ((float2 *)gmem_row_ptr)[lane_id];
    }
  }

  uint idx = lane_id + num_cols;
  if (idx < num_cols_after_padding) {
    for (uint row = 0; row < num_rows; row++) {
      half *smem_row_ptr = &smem_in[row * input_stride];
      smem_row_ptr[idx] = __float2half(0);
    }
  }

  half4 zeros;
#ifdef __HIP_PLATFORM_HCC__
  zeros.vals[0] = __float2half2_rn(0);
  zeros.vals[1] = __float2half2_rn(0);
#else
  half4 zeros;
  zeros.vals[0].x = __float2half(0);
  zeros.vals[0].y = __float2half(0);
  zeros.vals[1].x = __float2half(0);
  zeros.vals[1].y = __float2half(0);
#endif

  if (lane_id < (num_cols_after_padding >> 2)) {
#pragma unroll 2
    for (uint row = num_rows; row < num_rows_after_padding; row++) {
      half *smem_row_ptr = &smem_in[row * input_stride];
      ((half4 *)smem_row_ptr)[lane_id] = zeros;
    }
  }

  syncwarp();

  bmmBwdKernel<TILE_DIM, ROW_TILES_PER_STEP, TILE_DIM_LOG_2, __half>(
      smem_in, smem_temp, smem_out, gmem_grad, num_rows, num_cols, num_col_steps,
      input_stride, interaction_ugrad_2D_stride, lane_id);

  if (lane_id < (num_cols >> 2)) {
    ((float2 *)gmem_mlp_grad)[lane_id] = ((float2 *)gmem_ugrad)[lane_id];
  }
}

#ifdef __HIP_PLATFORM_HCC__
template <uint WARPS_PER_BLOCK,
          uint THREADBLOCK_SIZE,
          uint ROW_TILES_PER_STEP,
          uint COL_TILES_PER_STEP,
          uint WARP_SIZE,
          uint WARP_SIZE_LOG_2,
          uint TILE_DIM,
          uint TILE_DIM_LOG_2>
__launch_bounds__(THREADBLOCK_SIZE) __global__
    void dotBasedInteractBwdKernelNonAligned(const float *__restrict input,
                                             const float *__restrict upstream_grad,
                                             float *__restrict grad,
                                             float *__restrict bottom_mlp_grad,
                                             uint batch_size,
                                             uint num_rows,
                                             uint num_cols,
                                             uint num_rows_after_padding,
                                             uint num_cols_after_padding,
                                             uint sample_size,
                                             uint interaction_ugrad_size,
                                             uint interaction_ugrad_size_with_padding,
                                             uint interaction_ugrad_2D_size_elems,
                                             uint interaction_ugrad_2D_stride,
                                             uint input_size_elems,
                                             uint input_stride,
                                             uint num_row_steps,
                                             uint num_col_steps,
                                             uint row_tiles_per_step,
                                             uint shared_mem_per_warp_size_byte) {
  extern __shared__ float shmem_bwd_na_f[];
  uint warp_id = (threadIdx.x >> WARP_SIZE_LOG_2);
  uint sample_id = blockIdx.x * WARPS_PER_BLOCK + warp_id;
  if (sample_id >= batch_size) {
    return;
  }
  uint lane_id = threadIdx.x & (WARP_SIZE - 1);
  // ">> 2" to convert to float pointer
  uint smem_warp_offset = warp_id * (shared_mem_per_warp_size_byte >> 2);

  float *smem_in = shmem_bwd_na_f + smem_warp_offset;
  float *smem_temp = smem_in + input_size_elems;
  float *smem_out = smem_temp;

  // Global memory pointers for the current sample
  // Input
  uint gmem_input_sample_offset = sample_id * sample_size;
  const float *gmem_input = &input[gmem_input_sample_offset];

  // Interaction Gradient
  const uint &gmem_grad_sample_offset = gmem_input_sample_offset;
  float *gmem_grad = &grad[gmem_grad_sample_offset];

  // Bottom MLP gradient
  float *gmem_mlp_grad = &bottom_mlp_grad[sample_id * num_cols];

  // Upstream gradient vector
  uint gmem_ugrad_sample_offset = sample_id * (num_cols + interaction_ugrad_size_with_padding);
  const float *gmem_ugrad = &upstream_grad[gmem_ugrad_sample_offset];

  // Upstream gradient vector for interactions
  const float *gmem_ugrad_interactions = &gmem_ugrad[num_cols];

  // upstream grad -> shared memory (place in input section temporarily)
#pragma unroll
  for (uint idx = lane_id; idx < interaction_ugrad_size; idx += WARP_SIZE) {
    smem_in[idx] = gmem_ugrad_interactions[idx];
  }

  syncwarp();

  // Form the 2D ugrad matrix.
  trilBwdKernel(smem_in, smem_temp, 0.0f, num_rows, num_rows_after_padding,
      interaction_ugrad_2D_stride, lane_id);

  syncwarp();

  // Input -> Shared Memory

  for (uint row = 0; row < num_rows; row++) {
    float *smem_row_ptr = &smem_in[row * input_stride];
    const float *gmem_row_ptr = &gmem_input[row * num_cols];
    for (uint idx = lane_id; idx < num_cols; idx += WARP_SIZE) {
      smem_row_ptr[idx] = gmem_row_ptr[idx];
    }
    uint idx = lane_id + num_cols;
    if (idx < num_cols_after_padding) {
      smem_row_ptr[idx] = 0;
    }
  }

#pragma unroll 2
  for (uint row = num_rows; row < num_rows_after_padding; row++) {
    float *smem_row_ptr = &smem_in[row * input_stride];
    for (uint idx = lane_id; idx < num_cols_after_padding; idx += WARP_SIZE) {
      smem_row_ptr[idx] = 0;
    }
  }

  syncwarp();

  bmmBwdKernel<TILE_DIM, ROW_TILES_PER_STEP, TILE_DIM_LOG_2, float>(
      smem_in, smem_temp, smem_out, gmem_grad, num_rows, num_cols, num_col_steps,
      input_stride, interaction_ugrad_2D_stride, lane_id);

  for (uint idx = lane_id; idx < num_cols; idx += WARP_SIZE) {
    gmem_mlp_grad[idx] = gmem_ugrad[idx];
  }
}

template <uint WARPS_PER_BLOCK,
          uint THREADBLOCK_SIZE,
          uint ROW_TILES_PER_STEP,
          uint COL_TILES_PER_STEP,
          uint WARP_SIZE,
          uint WARP_SIZE_LOG_2,
          uint TILE_DIM,
          uint TILE_DIM_LOG_2>
__launch_bounds__(THREADBLOCK_SIZE) __global__
  void dotBasedInteractBwdKernel(const float *__restrict input,
                                 const float *__restrict upstream_grad,
                                 float *__restrict grad,
                                 float *__restrict bottom_mlp_grad,
                                 uint batch_size,
                                 uint num_rows,
                                 uint num_cols,
                                 uint num_rows_after_padding,
                                 uint num_cols_after_padding,
                                 uint sample_size,
                                 uint interaction_ugrad_size,
                                 uint interaction_ugrad_size_with_padding,
                                 uint interaction_ugrad_2D_size_elems,
                                 uint interaction_ugrad_2D_stride,
                                 uint input_size_elems,
                                 uint input_stride,
                                 uint num_row_steps,
                                 uint num_col_steps,
                                 uint row_tiles_per_step,
                                 uint shared_mem_per_warp_size_byte) {
  extern __shared__ float shmem_bwd_f[];
  uint warp_id = (threadIdx.x >> WARP_SIZE_LOG_2);
  uint sample_id = blockIdx.x * WARPS_PER_BLOCK + warp_id;
  if (sample_id >= batch_size) {
    return;
  }
  uint lane_id = threadIdx.x & (WARP_SIZE - 1);
  // ">> 2" to convert to float pointer
  uint smem_warp_offset = warp_id * (shared_mem_per_warp_size_byte >> 2);

  float *smem_in = shmem_bwd_f + smem_warp_offset;
  float *smem_temp = smem_in + input_size_elems;
  float *smem_out = smem_temp;

  // Global memory pointers for the current sample
  // Input
  uint gmem_input_sample_offset = sample_id * sample_size;
  const float *gmem_input = &input[gmem_input_sample_offset];

  // Interaction Gradient
  const uint &gmem_grad_sample_offset = gmem_input_sample_offset;
  float *gmem_grad = &grad[gmem_grad_sample_offset];

  // Bottom MLP gradient
  float *gmem_mlp_grad = &bottom_mlp_grad[sample_id * num_cols];

  // Upstream gradient vector
  uint gmem_ugrad_sample_offset = sample_id * (num_cols + interaction_ugrad_size_with_padding);
  const float *gmem_ugrad = &upstream_grad[gmem_ugrad_sample_offset];

  // Upstream gradient vector for interactions
  const float *gmem_ugrad_interactions = &gmem_ugrad[num_cols];

  // upstream grad -> shared memory (place in input section temporarily)
#pragma unroll
  for (uint idx = lane_id; idx < (interaction_ugrad_size >> 2); idx += WARP_SIZE) {
    ((float4 *)smem_in)[idx] = ((float4 *)gmem_ugrad_interactions)[idx];
  }
  uint offset = (interaction_ugrad_size >> 2) << 2;
  for (uint idx = lane_id + offset; idx < interaction_ugrad_size; idx += WARP_SIZE) {
    smem_in[idx] = gmem_ugrad_interactions[idx];
  }

  syncwarp();

  // Form the 2D ugrad matrix.
  trilBwdKernel(smem_in, smem_temp, 0.0f, num_rows, num_rows_after_padding,
      interaction_ugrad_2D_stride, lane_id);

  syncwarp();

  // Input -> Shared Memory
  if (lane_id < (num_cols >> 1)) {
    for (uint row = 0; row < num_rows; row++) {
      float *smem_row_ptr = &smem_in[row * input_stride];
      const float *gmem_row_ptr = &gmem_input[row * num_cols];
      ((float2 *)smem_row_ptr)[lane_id] = ((float2 *)gmem_row_ptr)[lane_id];
    }
  }

  uint idx = lane_id + num_cols;
  if (idx < num_cols_after_padding) {
    for (uint row = 0; row < num_rows; row++) {
      float *smem_row_ptr = &smem_in[row * input_stride];
      smem_row_ptr[idx] = 0;
    }
  }

  float4 zeros;
  zeros.data[0] = 0;
  zeros.data[1] = 0;
  zeros.data[2] = 0;
  zeros.data[3] = 0;
  if (lane_id < (num_cols_after_padding >> 2)) {
#pragma unroll 2
    for (uint row = num_rows; row < num_rows_after_padding; row++) {
      float *smem_row_ptr = &smem_in[row * input_stride];
      ((float4 *)smem_row_ptr)[lane_id] = zeros;
    }
  }

  syncwarp();

  bmmBwdKernel<TILE_DIM, ROW_TILES_PER_STEP, TILE_DIM_LOG_2, float>(
      smem_in, smem_temp, smem_out, gmem_grad, num_rows, num_cols, num_col_steps,
      input_stride, interaction_ugrad_2D_stride, lane_id);

  if (lane_id < (num_cols >> 1)) {
    ((float2 *)gmem_mlp_grad)[lane_id] = ((float2 *)gmem_ugrad)[lane_id];
  }
}
#endif // __HIP_PLATFORM_HCC__

inline void dotBasedInteractFwd(const void *input,
                                const void *bottom_mlp_output,
                                void *output,
                                uint batch_size,
                                uint num_rows,
                                uint num_cols,
                                uint pad,
                                bool amp_train) {
#ifdef __HIP_PLATFORM_HCC__
  const uint kWarpSize = 64;
#else
  const uint kWarpSize = 32;
#endif
  const uint kWarpSizeLog2 = dlrm_dot::Log2<kWarpSize>::value;
  const uint kTileDim = 16;
  const uint kTileDimLog2 = dlrm_dot::Log2<kTileDim>::value;
  const uint warps_per_threadblock = 128 / kWarpSize;
  const uint threadblock_size = warps_per_threadblock * kWarpSize;
  const uint kRowTilesPerStep = 2;
  const uint kColTilesPerStep = 1;

  // num tiles
  uint num_row_tiles = (num_rows + kTileDim - 1) >> kTileDimLog2;
  uint num_col_tiles = (num_cols + kTileDim - 1) >> kTileDimLog2;

  // number of rows and columns after padding
  uint num_rows_after_padding = kTileDim << 1;
  uint num_cols_after_padding = num_col_tiles << kTileDimLog2;

  uint num_row_steps = num_row_tiles / kRowTilesPerStep;
  uint num_col_steps = num_col_tiles / kColTilesPerStep;

  const uint K_BLOCKS = 8;
  const uint M_BLOCKS = 2;
  const uint SKEW_HALF = ((K_BLOCKS % 2) == 0) ? 8 : 0;
  const uint SMEM_STRIDE = (K_BLOCKS * 16 + SKEW_HALF);
  // multiple of 2 to guarantee 256-bit alignment for start of the row, at least 16 to safeload a tile
  const uint smem_rows_per_warp = M_BLOCKS << 4;
  const uint smem_elems_per_warp_mat = smem_rows_per_warp * SMEM_STRIDE;
  const uint SKEW_HALF_ACC = ((M_BLOCKS % 2) == 0) ? 8 : 0;
  const uint SMEM_STRIDE_ACC = (M_BLOCKS * 16 + SKEW_HALF_ACC);
  const uint smem_elems_per_warp_acc = M_BLOCKS * kTileDim * SMEM_STRIDE_ACC * 2;  // output in FP32
  const uint smem_elems_per_warp =
      (smem_elems_per_warp_mat > smem_elems_per_warp_acc) ? smem_elems_per_warp_mat : smem_elems_per_warp_acc;
  uint output_size = num_cols + (num_rows * (num_rows - 1) >> 1) + pad;

  bool float4_predicate = !((num_cols & 7) || (output_size & 7));

  // aligned case
  if (float4_predicate) {
    if (amp_train) {
      dotBasedInteractFwdKernel<warps_per_threadblock,
                                threadblock_size,
                                M_BLOCKS,
                                K_BLOCKS,
                                SMEM_STRIDE,
                                SMEM_STRIDE_ACC,
                                kWarpSize,
                                kWarpSizeLog2,
                                kTileDim,
                                kTileDimLog2>
          <<<(batch_size + warps_per_threadblock - 1) / warps_per_threadblock,
             threadblock_size,
             warps_per_threadblock * smem_elems_per_warp * sizeof(__half)>>>((const __half *)input,
                                                                             (half *)output,
                                                                             batch_size,
                                                                             num_rows,
                                                                             num_cols,
                                                                             num_rows_after_padding,
                                                                             num_cols_after_padding,
                                                                             smem_elems_per_warp,
                                                                             smem_rows_per_warp,
                                                                             output_size,
                                                                             num_row_steps,
                                                                             num_col_steps,
                                                                             pad);
    }
    else {
#ifdef __HIP_PLATFORM_HCC__
      dotBasedInteractFwdKernel<warps_per_threadblock,
                                threadblock_size,
                                M_BLOCKS,
                                K_BLOCKS,
                                SMEM_STRIDE,
                                SMEM_STRIDE_ACC,
                                kWarpSize,
                                kWarpSizeLog2,
                                kTileDim,
                                kTileDimLog2>
          <<<(batch_size + warps_per_threadblock - 1) / warps_per_threadblock,
             threadblock_size,
             warps_per_threadblock * smem_elems_per_warp * sizeof(float)>>>((const float *)input,
                                                                            (float *)output,
                                                                            batch_size,
                                                                            num_rows,
                                                                            num_cols,
                                                                            num_rows_after_padding,
                                                                            num_cols_after_padding,
                                                                            smem_elems_per_warp,
                                                                            smem_rows_per_warp,
                                                                            output_size,
                                                                            num_row_steps,
                                                                            num_col_steps,
                                                                            pad);
#else
      throw std::invalid_argument("CUDA WMMA does not support FP32.");
#endif
    }
  } else {
    if (amp_train) {
      dotBasedInteractFwdKernelNonAligned<warps_per_threadblock,
                                          threadblock_size,
                                          M_BLOCKS,
                                          K_BLOCKS,
                                          SMEM_STRIDE,
                                          SMEM_STRIDE_ACC,
                                          kWarpSize,
                                          kWarpSizeLog2,
                                          kTileDim,
                                          kTileDimLog2>
          <<<(batch_size + warps_per_threadblock - 1) / warps_per_threadblock,
             threadblock_size,
             warps_per_threadblock * smem_elems_per_warp * sizeof(__half)>>>((const __half *)input,
                                                                             (half *)output,
                                                                             batch_size,
                                                                             num_rows,
                                                                             num_cols,
                                                                             num_rows_after_padding,
                                                                             num_cols_after_padding,
                                                                             smem_elems_per_warp,
                                                                             smem_rows_per_warp,
                                                                             output_size,
                                                                             num_row_steps,
                                                                             num_col_steps,
                                                                             pad);
    }
    else {
#ifdef __HIP_PLATFORM_HCC__
      dotBasedInteractFwdKernelNonAligned<warps_per_threadblock,
                                          threadblock_size,
                                          M_BLOCKS,
                                          K_BLOCKS,
                                          SMEM_STRIDE,
                                          SMEM_STRIDE_ACC,
                                          kWarpSize,
                                          kWarpSizeLog2,
                                          kTileDim,
                                          kTileDimLog2>
          <<<(batch_size + warps_per_threadblock - 1) / warps_per_threadblock,
             threadblock_size,
             warps_per_threadblock * smem_elems_per_warp * sizeof(float)>>>((const float *)input,
                                                                            (float *)output,
                                                                            batch_size,
                                                                            num_rows,
                                                                            num_cols,
                                                                            num_rows_after_padding,
                                                                            num_cols_after_padding,
                                                                            smem_elems_per_warp,
                                                                            smem_rows_per_warp,
                                                                            output_size,
                                                                            num_row_steps,
                                                                            num_col_steps,
                                                                            pad);
#else
      throw std::invalid_argument("CUDA WMMA does not support FP32.");
#endif
    }
  }
}

inline void dotBasedInteractBwd(void *input,
                                void *upstream_grad,
                                void *grad,
                                void *bottom_mlp_grad,
                                uint batch_size,
                                uint num_rows,
                                uint num_cols,
                                uint pad,
                                bool amp_train) {
#ifdef __HIP_PLATFORM_HCC__
  const uint kWarpSize = 64;
#else
  const uint kWarpSize = 32;
#endif
  const uint kWarpSizeLog2 = dlrm_dot::Log2<kWarpSize>::value;
  const uint kTileDim = 16;
  const uint kTileDimLog2 = dlrm_dot::Log2<kTileDim>::value;
  const uint mem_skew_size = 8;
  const uint kWarpsPerBlock = 128 / kWarpSize;
  const uint kWarpsPerBlockLog2 = dlrm_dot::Log2<kWarpsPerBlock>::value;
  const uint kNumThreads = kWarpsPerBlock * kWarpSize;
  const uint kRowTilesPerStep = 2;
  const uint kColTilesPerStep = 1;

  uint input_dbytes = amp_train ? sizeof(half) : sizeof(float);

  uint row_tiles_per_step = num_rows > kTileDim ? kRowTilesPerStep : 1;

  // num tiles
  uint num_row_tiles = (num_rows + kTileDim - 1) >> kTileDimLog2;
  uint num_col_tiles = (num_cols + kTileDim - 1) >> kTileDimLog2;

  // number of rows and columns after padding
  uint num_rows_after_padding = kTileDim << 1;
  uint num_cols_after_padding = num_col_tiles << kTileDimLog2;

  // 2D ugrad size and stride
  uint interaction_ugrad_2D_stride = num_rows_after_padding + mem_skew_size;
  uint interaction_ugrad_2D_size_elems = num_rows_after_padding * interaction_ugrad_2D_stride;
  uint interaction_ugrad_2D_size_bytes = interaction_ugrad_2D_size_elems * input_dbytes;

  // 1D ugrad size
  uint interaction_ugrad_size = num_rows * (num_rows - 1) >> 1;
  uint interaction_ugrad_size_with_padding = interaction_ugrad_size + pad;

  // in_out place size and stride
  uint input_stride = num_cols_after_padding + mem_skew_size;
  uint input_size_elems = num_rows_after_padding * input_stride;
  uint input_size_bytes = input_size_elems * input_dbytes;

  // sample size
  uint sample_size = num_rows * num_cols;

  // output size
  uint output_size_elems = kTileDim * kTileDim * kRowTilesPerStep * kColTilesPerStep;
  uint output_size_bytes = output_size_elems * sizeof(float);

  // staging area size
  uint staging_area_size_bytes =
      output_size_bytes > interaction_ugrad_2D_size_bytes ? output_size_bytes : interaction_ugrad_2D_size_bytes;

  // Shared memory size
  uint shared_mem_per_warp_size_byte = input_size_bytes + staging_area_size_bytes;
  uint shared_mem_size_bytes = kWarpsPerBlock * shared_mem_per_warp_size_byte;

  uint num_blocks = (batch_size + kWarpsPerBlock - 1) >> kWarpsPerBlockLog2;
  uint num_row_steps = num_row_tiles / row_tiles_per_step;
  uint num_col_steps = num_col_tiles / kColTilesPerStep;

  bool float4_predicate = !((interaction_ugrad_size_with_padding & 7) || (num_cols & 7));
  if (float4_predicate) {
    if (amp_train) {
      dotBasedInteractBwdKernel<kWarpsPerBlock,
                                kNumThreads,
                                kRowTilesPerStep,
                                kColTilesPerStep,
                                kWarpSize,
                                kWarpSizeLog2,
                                kTileDim,
                                kTileDimLog2>
          <<<num_blocks, kNumThreads, shared_mem_size_bytes>>>((const half *)input,
                                                               (const half *)upstream_grad,
                                                               (half *)grad,
                                                               (half *)bottom_mlp_grad,
                                                               batch_size,
                                                               num_rows,
                                                               num_cols,
                                                               num_rows_after_padding,
                                                               num_cols_after_padding,
                                                               sample_size,
                                                               interaction_ugrad_size,
                                                               interaction_ugrad_size_with_padding,
                                                               interaction_ugrad_2D_size_elems,
                                                               interaction_ugrad_2D_stride,
                                                               input_size_elems,
                                                               input_stride,
                                                               num_row_steps,
                                                               num_col_steps,
                                                               row_tiles_per_step,
                                                               shared_mem_per_warp_size_byte);
    }
    else {
#ifdef __HIP_PLATFORM_HCC__
      dotBasedInteractBwdKernel<kWarpsPerBlock,
                                kNumThreads,
                                kRowTilesPerStep,
                                kColTilesPerStep,
                                kWarpSize,
                                kWarpSizeLog2,
                                kTileDim,
                                kTileDimLog2>
          <<<num_blocks, kNumThreads, shared_mem_size_bytes>>>((const float *)input,
                                                               (const float *)upstream_grad,
                                                               (float *)grad,
                                                               (float *)bottom_mlp_grad,
                                                               batch_size,
                                                               num_rows,
                                                               num_cols,
                                                               num_rows_after_padding,
                                                               num_cols_after_padding,
                                                               sample_size,
                                                               interaction_ugrad_size,
                                                               interaction_ugrad_size_with_padding,
                                                               interaction_ugrad_2D_size_elems,
                                                               interaction_ugrad_2D_stride,
                                                               input_size_elems,
                                                               input_stride,
                                                               num_row_steps,
                                                               num_col_steps,
                                                               row_tiles_per_step,
                                                               shared_mem_per_warp_size_byte);
#else
      throw std::invalid_argument("CUDA WMMA does not support FP32.");
#endif
    }
  } else {
    if (amp_train) {
      dotBasedInteractBwdKernelNonAligned<kWarpsPerBlock,
                                          kNumThreads,
                                          kRowTilesPerStep,
                                          kColTilesPerStep,
                                          kWarpSize,
                                          kWarpSizeLog2,
                                          kTileDim,
                                          kTileDimLog2>
          <<<num_blocks, kNumThreads, shared_mem_size_bytes>>>((const half *)input,
                                                               (const half *)upstream_grad,
                                                               (half *)grad,
                                                               (half *)bottom_mlp_grad,
                                                               batch_size,
                                                               num_rows,
                                                               num_cols,
                                                               num_rows_after_padding,
                                                               num_cols_after_padding,
                                                               sample_size,
                                                               interaction_ugrad_size,
                                                               interaction_ugrad_size_with_padding,
                                                               interaction_ugrad_2D_size_elems,
                                                               interaction_ugrad_2D_stride,
                                                               input_size_elems,
                                                               input_stride,
                                                               num_row_steps,
                                                               num_col_steps,
                                                               row_tiles_per_step,
                                                               shared_mem_per_warp_size_byte);
    }
    else {
#ifdef __HIP_PLATFORM_HCC__
      dotBasedInteractBwdKernelNonAligned<kWarpsPerBlock,
                                          kNumThreads,
                                          kRowTilesPerStep,
                                          kColTilesPerStep,
                                          kWarpSize,
                                          kWarpSizeLog2,
                                          kTileDim,
                                          kTileDimLog2>
          <<<num_blocks, kNumThreads, shared_mem_size_bytes>>>((const float *)input,
                                                               (const float *)upstream_grad,
                                                               (float *)grad,
                                                               (float *)bottom_mlp_grad,
                                                               batch_size,
                                                               num_rows,
                                                               num_cols,
                                                               num_rows_after_padding,
                                                               num_cols_after_padding,
                                                               sample_size,
                                                               interaction_ugrad_size,
                                                               interaction_ugrad_size_with_padding,
                                                               interaction_ugrad_2D_size_elems,
                                                               interaction_ugrad_2D_stride,
                                                               input_size_elems,
                                                               input_stride,
                                                               num_row_steps,
                                                               num_col_steps,
                                                               row_tiles_per_step,
                                                               shared_mem_per_warp_size_byte);
#else
      throw std::invalid_argument("CUDA WMMA does not support FP32.");
#endif
    }
  }
}
#endif // USE_WMMA
